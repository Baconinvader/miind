#include "hip/hip_runtime.h"
// Copyright (c) 2005 - 2015 Marc de Kamps
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without modification, are permitted provided that the following conditions are met:
//
//    * Redistributions of source code must retain the above copyright notice, this list of conditions and the following disclaimer.
//    * Redistributions in binary form must reproduce the above copyright notice, this list of conditions and the following disclaimer in the documentation
//      and/or other materials provided with the distribution.
//    * Neither the name of the copyright holder nor the names of its contributors may be used to endorse or promote products derived from this software
//      without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
// WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY
// DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF
// USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING
// NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//
//      If you use this software in work leading to a scientific publication, you should include a reference there to
//      the 'currently valid reference', which can be found at http://miind.sourceforge.net

#include <iostream>
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cmath>
#include "CudaEuler.cuh"
#include "CSRAdapter.cuh"

using namespace CudaTwoDLib;

const fptype TOLERANCE = 1e-9;


#define checkCudaErrors(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

void CSRAdapter::FillMatrixMaps(const std::vector<TwoDLib::CSRMatrix>& vecmat)
{
    for (inttype m = 0; m < vecmat.size(); m++)
    {
        _nval[m] = vecmat[m].Val().size();
        checkCudaErrors(hipMalloc((fptype**)&_val[m], _nval[m] * sizeof(fptype)));
        // dont't depend on Val() being of fptype
        std::vector<fptype> vecval;
        for (fptype val : vecmat[m].Val())
            vecval.push_back(val);
        checkCudaErrors(hipMemcpy(_val[m], &vecval[0], sizeof(fptype) * _nval[m], hipMemcpyHostToDevice));

        _nia[m] = vecmat[m].Ia().size();
        checkCudaErrors(hipMalloc((inttype**)&_ia[m], _nia[m] * sizeof(inttype)));
        std::vector<inttype> vecia;
        for (inttype ia : vecmat[m].Ia())
            vecia.push_back(ia);
        checkCudaErrors(hipMemcpy(_ia[m], &vecia[0], sizeof(inttype) * _nia[m], hipMemcpyHostToDevice));


        _nja[m] = vecmat[m].Ja().size();
        checkCudaErrors(hipMalloc((inttype**)&_ja[m], _nja[m] * sizeof(inttype)));
        std::vector<inttype> vecja;
        for (inttype ja : vecmat[m].Ja())
            vecja.push_back(ja);
        checkCudaErrors(hipMemcpy(_ja[m], &vecja[0], sizeof(inttype) * _nja[m], hipMemcpyHostToDevice));
    }
}

void CSRAdapter::FillForwardMatrixMaps(const std::vector<TwoDLib::CSRMatrix>& vecmat)
{
    for (inttype m = 0; m < vecmat.size(); m++)
    {
        checkCudaErrors(hipMalloc((fptype**)&_forward_val[m], vecmat[m].ForwardVal().size() * sizeof(fptype)));
        // dont't depend on Val() being of fptype
        std::vector<fptype> vecval;
        for (fptype val : vecmat[m].ForwardVal())
            vecval.push_back(val);
        checkCudaErrors(hipMemcpy(_forward_val[m], &vecval[0], sizeof(fptype) * vecmat[m].ForwardVal().size(), hipMemcpyHostToDevice));

        checkCudaErrors(hipMalloc((inttype**)&_forward_ia[m], vecmat[m].ForwardIa().size() * sizeof(inttype)));
        std::vector<inttype> vecia;
        for (inttype ia : vecmat[m].ForwardIa())
            vecia.push_back(ia);
        checkCudaErrors(hipMemcpy(_forward_ia[m], &vecia[0], sizeof(inttype) * vecmat[m].ForwardIa().size(), hipMemcpyHostToDevice));

        checkCudaErrors(hipMalloc((inttype**)&_forward_ja[m], vecmat[m].ForwardJa().size() * sizeof(inttype)));
        std::vector<inttype> vecja;
        for (inttype ja : vecmat[m].ForwardJa())
            vecja.push_back(ja);
        checkCudaErrors(hipMemcpy(_forward_ja[m], &vecja[0], sizeof(inttype) * vecmat[m].ForwardJa().size(), hipMemcpyHostToDevice));
    }
}

void CSRAdapter::InitializeStaticGridEfficacies(const std::vector<inttype>& vecindex, const std::vector<fptype>& efficacy, const std::vector<fptype>& cell_width, const std::vector<inttype>& grid_efficacy_offset) {
    _nr_grid_connections = efficacy.size();
    for (inttype m = 0; m < efficacy.size(); m++)
    {
        checkCudaErrors(hipMalloc((fptype**)&_goes[m], _nr_rows[vecindex[m]] * sizeof(fptype)));
        checkCudaErrors(hipMalloc((fptype**)&_stays[m], _nr_rows[vecindex[m]] * sizeof(fptype)));
        checkCudaErrors(hipMalloc((inttype**)&_offset1s[m], _nr_rows[vecindex[m]] * sizeof(inttype)));
        checkCudaErrors(hipMalloc((inttype**)&_offset2s[m], _nr_rows[vecindex[m]] * sizeof(inttype)));

        inttype numBlocks = (_nr_rows[vecindex[m]] + _blockSize - 1) / _blockSize;

        CudaCalculateGridEfficacies << <numBlocks, _blockSize >> > (_nr_rows[vecindex[m]],
            efficacy[m], cell_width[m], grid_efficacy_offset[m],
            _stays[m], _goes[m], _offset1s[m], _offset2s[m]);
    }
}

void CSRAdapter::InitializeStaticGridCellEfficacies(const std::vector<inttype>& vecindex, const std::vector<std::vector<fptype>>& vals, const std::vector<fptype>& cell_width, const std::vector<inttype>& grid_efficacy_offset) {
    _nr_grid_connections = vals.size();
    for (inttype m = 0; m < vals.size(); m++)
    {
        checkCudaErrors(hipMalloc((fptype**)&_goes[m], _nr_rows[vecindex[m]] * sizeof(fptype)));
        checkCudaErrors(hipMalloc((fptype**)&_stays[m], _nr_rows[vecindex[m]] * sizeof(fptype)));
        checkCudaErrors(hipMalloc((inttype**)&_offset1s[m], _nr_rows[vecindex[m]] * sizeof(inttype)));
        checkCudaErrors(hipMalloc((inttype**)&_offset2s[m], _nr_rows[vecindex[m]] * sizeof(inttype)));
        checkCudaErrors(hipMalloc((fptype**)&_cell_vals[m], vals[m].size() * sizeof(fptype)));
        checkCudaErrors(hipMemcpy(_cell_vals[m], &vals[m][0], vals[m].size() * sizeof(fptype), hipMemcpyHostToDevice));

        inttype numBlocks = (_nr_rows[vecindex[m]] + _blockSize - 1) / _blockSize;

        CudaCalculateGridCellEfficacies << <numBlocks, _blockSize >> > (_nr_rows[vecindex[m]],
            _cell_vals[m], cell_width[m], grid_efficacy_offset[m],
            _stays[m], _goes[m], _offset1s[m], _offset2s[m], _offsets[vecindex[m]]);
    }
}

void CSRAdapter::InitializeStaticGridCellCsrNd(const std::vector<inttype>& vecindex, const std::vector<TwoDLib::CSRMatrix>& mats) {

    _nr_grid_connections = mats.size();

    for (inttype m = 0; m < _nr_grid_connections; m++) {

        // yikes, convert the array of doubles to fptypes
        std::vector<fptype> d2f(mats[m].Val().size());
        for (int f = 0; f < mats[m].Val().size(); f++)
            d2f[f] = (fptype)mats[m].Val()[f];

        std::vector<fptype> fd2f(mats[m].ForwardVal().size());
        for (int f = 0; f < mats[m].ForwardVal().size(); f++)
            fd2f[f] = (fptype)mats[m].ForwardVal()[f];

        checkCudaErrors(hipMalloc((fptype**)&_grid_val[m], d2f.size() * sizeof(fptype)));
        checkCudaErrors(hipMalloc((inttype**)&_grid_ia[m], mats[m].Ia().size() * sizeof(inttype)));
        checkCudaErrors(hipMalloc((inttype**)&_grid_ja[m], mats[m].Ja().size() * sizeof(inttype)));
        checkCudaErrors(hipMalloc((fptype**)&_grid_forward_val[m], fd2f.size() * sizeof(fptype)));
        checkCudaErrors(hipMalloc((inttype**)&_grid_forward_ia[m], mats[m].ForwardIa().size() * sizeof(inttype)));
        checkCudaErrors(hipMalloc((inttype**)&_grid_forward_ja[m], mats[m].ForwardJa().size() * sizeof(inttype)));


        checkCudaErrors(hipMemcpy(_grid_val[m], &d2f[0], d2f.size() * sizeof(fptype), hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(_grid_ia[m], &mats[m].Ia()[0], mats[m].Ia().size() * sizeof(inttype), hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(_grid_ja[m], &mats[m].Ja()[0], mats[m].Ja().size() * sizeof(inttype), hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(_grid_forward_val[m], &fd2f[0], fd2f.size() * sizeof(fptype), hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(_grid_forward_ia[m], &mats[m].ForwardIa()[0], mats[m].ForwardIa().size() * sizeof(inttype), hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(_grid_forward_ja[m], &mats[m].ForwardJa()[0], mats[m].ForwardJa().size() * sizeof(inttype), hipMemcpyHostToDevice));
    }

}

void CSRAdapter::InitializeStaticGridConductanceEfficacies(const std::vector<inttype>& vecindex,
    const std::vector<fptype>& efficacy, const std::vector<fptype>& cell_widths, const std::vector<inttype>& cell_offsets, const std::vector<fptype>& rest_vs) {
    _nr_grid_connections = efficacy.size();

    checkCudaErrors(hipMalloc((fptype**)&_cell_vs, _group.getGroup().Vs().size() * sizeof(fptype)));

    std::vector<fptype> vecval;
    for (double val : _group.getGroup().Vs())
        vecval.push_back((fptype)val);

    checkCudaErrors(hipMemcpy(_cell_vs, &vecval[0], _group.getGroup().Vs().size() * sizeof(fptype), hipMemcpyHostToDevice));

    for (inttype m = 0; m < efficacy.size(); m++)
    {
        checkCudaErrors(hipMalloc((fptype**)&_goes[m], _nr_rows[vecindex[m]] * sizeof(fptype)));
        checkCudaErrors(hipMalloc((fptype**)&_stays[m], _nr_rows[vecindex[m]] * sizeof(fptype)));
        checkCudaErrors(hipMalloc((inttype**)&_offset1s[m], _nr_rows[vecindex[m]] * sizeof(inttype)));
        checkCudaErrors(hipMalloc((inttype**)&_offset2s[m], _nr_rows[vecindex[m]] * sizeof(inttype)));

        inttype numBlocks = (_nr_rows[vecindex[m]] + _blockSize - 1) / _blockSize;

        CudaCalculateGridEfficaciesWithConductance << <numBlocks, _blockSize >> > (_nr_rows[vecindex[m]],
            efficacy[m], cell_widths[m], cell_offsets[m], _cell_vs, rest_vs[m],
            _stays[m], _goes[m], _offset1s[m], _offset2s[m], _offsets[vecindex[m]]);
    }
}


void CSRAdapter::DeleteMatrixMaps()
{
    for (inttype m = 0; m < _nr_m; m++)
    {
        hipFree(_val[m]);
        hipFree(_ia[m]);
        hipFree(_ja[m]);
    }
}

void CSRAdapter::DeleteForwardMatrixMaps()
{
    for (inttype m = 0; m < _nr_m; m++)
    {
        hipFree(_forward_val[m]);
        hipFree(_forward_ia[m]);
        hipFree(_forward_ja[m]);
    }
}

inttype CSRAdapter::NumberIterations(const CudaOde2DSystemAdapter& group, fptype euler_timestep) const
{
    fptype tstep = group._group.MeshObjects()[0].TimeStep();
    for (const auto& mesh : group._group.MeshObjects())
        if (fabs(tstep - mesh.TimeStep()) > TOLERANCE) {
            std::cerr << "Not all meshes in this group have the same time step. " << tstep << " " << mesh.TimeStep() << " " << tstep - mesh.TimeStep() << std::endl;
            exit(0);
        }
    inttype  n_steps = static_cast<inttype>(std::round(tstep / euler_timestep));

    return n_steps;
}

void CSRAdapter::InspectMass(inttype i)
{
    std::vector<fptype> hostvec(_group._n);
    checkCudaErrors(hipMemcpy(&hostvec[0], _group._mass, sizeof(fptype) * _group._n, hipMemcpyDeviceToHost));
}

CSRAdapter::CSRAdapter(CudaOde2DSystemAdapter& group, const std::vector<TwoDLib::CSRMatrix>& vecmat,
    inttype nr_grid_connections, fptype euler_timestep,
    const std::vector<inttype>& vecmat_indexes, const std::vector<inttype>& grid_transforms) :
    _group(group),
    _euler_timestep(euler_timestep),
    _nr_iterations(NumberIterations(group, euler_timestep)),
    _nr_m(vecmat.size()),
    _nr_streams(vecmat.size()),
    _vecmats(vecmat_indexes),
    _grid_transforms(grid_transforms),
    _nval(std::vector<inttype>(vecmat.size())),
    _val(std::vector<fptype*>(vecmat.size())),
    _forward_val(std::vector<fptype*>(vecmat.size())),
    _nia(std::vector<inttype>(vecmat.size())),
    _ia(std::vector<inttype*>(vecmat.size())),
    _forward_ia(std::vector<inttype*>(vecmat.size())),
    _nja(std::vector<inttype>(vecmat.size())),
    _ja(std::vector<inttype*>(vecmat.size())),
    _forward_ja(std::vector<inttype*>(vecmat.size())),
    _offsets(this->Offsets(vecmat)),
    _nr_rows(this->NrRows(vecmat)),
    _goes(nr_grid_connections),
    _stays(nr_grid_connections),
    _offset1s(nr_grid_connections),
    _offset2s(nr_grid_connections),
    _cell_vals(nr_grid_connections),
    _grid_val(nr_grid_connections),
    _grid_ia(nr_grid_connections),
    _grid_ja(nr_grid_connections),
    _grid_forward_val(nr_grid_connections),
    _grid_forward_ia(nr_grid_connections),
    _grid_forward_ja(nr_grid_connections),
    _blockSize(256),
    _numBlocks((_group._n + _blockSize - 1) / _blockSize)
{
    this->FillMatrixMaps(vecmat);
    this->FillForwardMatrixMaps(vecmat);
    this->FillDerivative();
    this->CreateStreams();
    this->FillRandom();
    // Speed Testing for comparison - Izhikevich neurons on the GPU
#ifdef IZHIKEVICH_TEST
    this->FillIzhVectors();
#endif
}

// Speed Testing for comparison - Izhikevich neurons on the GPU
void CSRAdapter::FillIzhVectors() {
    checkCudaErrors(hipMalloc((fptype**)&_izh_vs, _group.NumObjects() * sizeof(fptype)));
    checkCudaErrors(hipMalloc((fptype**)&_izh_ws, _group.NumObjects() * sizeof(fptype)));
    checkCudaErrors(hipMalloc((fptype**)&_refract_times, _group.NumObjects() * sizeof(fptype)));

    std::vector<fptype> vs(_group.NumObjects(), -70.0);
    std::vector<fptype> ws(_group.NumObjects(), 0.0);
    std::vector<fptype> refract_times(_group.NumObjects(), -1.0);

    checkCudaErrors(hipMemcpy(_izh_vs, &vs[0], _group.NumObjects() * sizeof(fptype), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(_izh_ws, &ws[0], _group.NumObjects() * sizeof(fptype), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(_refract_times, &refract_times[0], _group.NumObjects() * sizeof(fptype), hipMemcpyHostToDevice));
}

// Speed Testing for comparison - Izhikevich neurons on the GPU
void CSRAdapter::IzhTest(inttype* spikes) {
    inttype numBlocks = (_group.NumObjects() + _blockSize - 1) / _blockSize;
    generatePoissonSpikes << <numBlocks, _blockSize >> > (_group.NumObjects(), 0, 5000 * 2, 0.0001, _random_poisson, _randomState);

    CudaSolveIzhikevichNeurons << <numBlocks, _blockSize >> > (_group.NumObjects(), _random_poisson, spikes, _izh_vs, _izh_ws, _refract_times, 0.0, 0.0001, _randomState);

}

CSRAdapter::CSRAdapter(CudaOde2DSystemAdapter& group, const std::vector<TwoDLib::CSRMatrix>& vecmat, fptype euler_timestep) :
    CSRAdapter(group, vecmat, vecmat.size(), euler_timestep,
        std::vector<inttype>(), std::vector<inttype>()) {
    for (unsigned int i = 0; i < vecmat.size(); i++)
        _vecmats.push_back(i);
}

CSRAdapter::~CSRAdapter()
{
#ifdef IZHIKEVICH_TEST
    free(_izh_vs);
    free(_izh_ws);
    free(_refract_times);
#endif

    hipFree(_cell_vs);

    for (inttype m = 0; m < _nr_grid_connections; m++) {
        hipFree(_grid_val[m]);
        hipFree(_grid_ia[m]);
        hipFree(_grid_ja[m]);
        hipFree(_grid_forward_val[m]);
        hipFree(_grid_forward_ia[m]);
        hipFree(_grid_forward_ja[m]);
        hipFree(_cell_vals[m]);
        hipFree(_goes[m]);
        hipFree(_stays[m]);
        hipFree(_offset1s[m]);
        hipFree(_offset2s[m]);
    }

    this->DeleteMatrixMaps();
    this->DeleteForwardMatrixMaps();
    this->DeleteDerivative();
    this->DeleteStreams();
    this->DeleteRandom();
}

void CSRAdapter::CreateStreams()
{
    _streams = (hipStream_t*)malloc(_nr_streams * sizeof(hipStream_t));
    for (int i = 0; i < _nr_streams; i++)
        hipStreamCreate(&_streams[i]);
}

void CSRAdapter::DeleteStreams()
{
    hipFree(_streams);
}

void CSRAdapter::FillDerivative()
{

    checkCudaErrors(hipMalloc((fptype**)&_dydt, _group._n * sizeof(fptype)));
}

void CSRAdapter::FillRandom()
{
    inttype numBlocks = (_group.NumObjects() + _blockSize - 1) / _blockSize;
    checkCudaErrors(hipMalloc((inttype**)&_random_poisson, _group.NumObjects() * sizeof(inttype)));
    checkCudaErrors(hipMalloc((void**)&_randomState, _blockSize * numBlocks * sizeof(hiprandState)));
}

void CSRAdapter::DeleteRandom()
{
    hipFree(_random_poisson);
    hipFree(_randomState);
}

void CSRAdapter::DeleteDerivative()
{
    hipFree(_dydt);
}

void CSRAdapter::ClearDerivative()
{

    inttype n = _group._n;
    CudaClearDerivative << <_numBlocks, _blockSize >> > (n, _dydt);

}

std::vector<inttype> CSRAdapter::NrRows(const std::vector<TwoDLib::CSRMatrix>& vecmat) const
{
    std::vector<inttype> vecret;
    for (inttype m = 0; m < vecmat.size(); m++)
        vecret.push_back(vecmat[m].NrRows());
    return vecret;
}

std::vector<inttype> CSRAdapter::Offsets(const std::vector<TwoDLib::CSRMatrix>& vecmat) const
{
    std::vector<inttype> vecret;
    for (inttype m : _group.getGroup().Offsets())
        vecret.push_back(m);
    return vecret;
}

void CSRAdapter::CalculateDerivative(const std::vector<fptype>& vecrates)
{
    for (inttype m : _vecmats)
    {
        // be careful to use this block size
        inttype numBlocks = (_nr_rows[m] + _blockSize - 1) / _blockSize;
        CudaCalculateDerivative << <numBlocks, _blockSize >> > (_nr_rows[m], vecrates[m], _dydt, _group._mass, _val[m], _ia[m], _ja[m], _group._map, _offsets[m]);
    }

}

void CSRAdapter::CalculateGridDerivative(const std::vector<inttype>& vecindex, const std::vector<fptype>& vecrates, const std::vector<fptype>& vecstays, const std::vector<fptype>& vecgoes, const std::vector<int>& vecoff1s, const std::vector<int>& vecoff2s)
{
    for (inttype m = 0; m < vecindex.size(); m++)
    {
        // be careful to use this block size
        inttype numBlocks = (_nr_rows[vecindex[m]] + _blockSize - 1) / _blockSize;
        CudaCalculateGridDerivative << <numBlocks, _blockSize, 0, _streams[vecindex[m]] >> > (_nr_rows[vecindex[m]], vecrates[m], vecstays[m], vecgoes[m], vecoff1s[m], vecoff2s[m], _dydt, _group._mass, _offsets[m]);
    }

    hipDeviceSynchronize();
}

void CSRAdapter::CalculateMeshGridDerivative(const std::vector<inttype>& vecindex,
    const std::vector<fptype>& vecrates, const std::vector<fptype>& vecstays,
    const std::vector<fptype>& vecgoes, const std::vector<int>& vecoff1s,
    const std::vector<int>& vecoff2s)
{

    for (inttype m = 0; m < vecstays.size(); m++)
    {
        // be careful to use this block size
        inttype numBlocks = (_nr_rows[vecindex[m]] + _blockSize - 1) / _blockSize;
        CudaCalculateGridDerivative << <numBlocks, _blockSize, 0, _streams[vecindex[m]] >> > (_nr_rows[vecindex[m]], vecrates[m], vecstays[m], vecgoes[m], vecoff1s[m], vecoff2s[m], _dydt, _group._mass, _offsets[vecindex[m]]);
    }

    for (int n = vecstays.size(); n < vecrates.size(); n++)
    {
        inttype mat_index = _grid_transforms.size() + (n - vecstays.size());
        // be careful to use this block size
        inttype numBlocks = (_nr_rows[mat_index] + _blockSize - 1) / _blockSize;
        CudaCalculateDerivative << <numBlocks, _blockSize, 0, _streams[vecindex[n]] >> > (_nr_rows[mat_index], vecrates[n], _dydt, _group._mass, _val[mat_index], _ia[mat_index], _ja[mat_index], _group._map, _offsets[mat_index]);
    }

    hipDeviceSynchronize();

}


void CSRAdapter::CalculateMeshGridDerivativeWithEfficacy(const std::vector<inttype>& vecindex, const std::vector<inttype>& in_vecindex,
    const std::vector<fptype>& vecrates)
{

    for (inttype m = 0; m < _nr_grid_connections; m++)
    {
        unsigned int mesh_m = vecindex[m];
        unsigned int in_mesh_m = in_vecindex[m];

        if (_group.getGroupObjects()[mesh_m] > 0) {
            continue;
        }

        // be careful to use this block size
        inttype numBlocks = (_nr_rows[vecindex[m]] + _blockSize - 1) / _blockSize;

        dim3 numBlocks2D;
        dim3 blockSize2D = dim3(_blockSize, 1);
        if (in_mesh_m < _group._kernels.size() && _group._kernels[in_mesh_m].size() > 0) {
            // kernel
            numBlocks2D = dim3(numBlocks, _group._kernels[in_mesh_m].size());
        }
        else {
            // no kernel
            numBlocks2D = dim3(numBlocks, 1);
        }

        CudaCalculateGridDerivativeCsrKernel << <numBlocks, _blockSize, 0, _streams[vecindex[m]] >> > (_nr_rows[vecindex[m]], vecrates[m], _dydt, _group._mass, _group._mass_histories, _group._host_mass_histories.size(),
            _grid_val[m], _grid_ia[m], _grid_ja[m], _offsets[mesh_m], _group._vec_vec_kernels[in_mesh_m], _group._kernels[in_mesh_m].size());


    }


    for (int n = _nr_grid_connections; n < vecrates.size(); n++)
    {
        inttype mat_index = _grid_transforms.size() + (n - _nr_grid_connections);

        unsigned int mesh_m = vecindex[n];
        unsigned int in_mesh_m = in_vecindex[n];

        if (_group.getGroupObjects()[mesh_m] > 0) {
            continue;
        }

        // be careful to use this block size
        inttype numBlocks = (_nr_rows[mat_index] + _blockSize - 1) / _blockSize;

        dim3 numBlocks2D;
        dim3 blockSize2D = dim3(_blockSize, 1);
        if (in_mesh_m < _group._kernels.size() && _group._kernels[in_mesh_m].size() > 0) {
            // kernel
            numBlocks2D = dim3(numBlocks, _group._kernels[in_mesh_m].size());
        }
        else {
            // no kernel
            numBlocks2D = dim3(numBlocks, 1);
        }

        CudaCalculateDerivativeKernel << < numBlocks2D, blockSize2D, 0, _streams[vecindex[n]] >> > (_nr_rows[mat_index], vecrates[n], _dydt, _group._mass, _group._mass_histories, _group._host_mass_histories.size(),
            _val[mat_index], _ia[mat_index], _ja[mat_index], _group._map, _offsets[mesh_m],
            _group._vec_vec_kernels[in_mesh_m], _group._kernels[in_mesh_m].size());

    }

    hipDeviceSynchronize();

}



void CSRAdapter::SingleTransformStep()
{

    for (inttype m : _grid_transforms)
    {
        if (_group._vec_num_objects[m] > 0)
            continue;

        // be careful to use this block size
        inttype numBlocks = (_nr_rows[m] + _blockSize - 1) / _blockSize;
        CudaSingleTransformStep << <numBlocks, _blockSize, 0, _streams[m] >> > (_nr_rows[m], _dydt, _group._mass, _val[m], _ia[m], _ja[m], _group._map, _offsets[m]);
    }

}

void CSRAdapter::SingleTransformStepFiniteSize()
{

    for (inttype m : _grid_transforms)
    {
        if (_group._vec_num_objects[m] == 0)
            continue;
        // be careful to use this block size
        inttype numBlocks = (_group._vec_num_objects[m] + _blockSize - 1) / _blockSize;
        CudaGridEvolveFiniteObjects << <numBlocks, _blockSize, 0, _streams[m] >> >
            (_group._vec_num_objects[m], _group._vec_num_object_offsets[m], _group._vec_objects_to_index, _group._vec_objects_refract_times,
                _forward_val[m], _forward_ia[m], _forward_ja[m],
                _offsets[m], _randomState);
    }
}


void CSRAdapter::AddDerivative()
{
    EulerStep << <_numBlocks, _blockSize >> > (_group._n, _dydt, _group._mass, _euler_timestep);
}

void CSRAdapter::AddDerivativeFull()
{
    EulerStep << <_numBlocks, _blockSize >> > (_group._n, _dydt, _group._mass, 1.0);
}

void CSRAdapter::setRandomSeeds(double seed) {


    inttype numBlocks = (_group.NumObjects() + _blockSize - 1) / _blockSize;

    initCurand << <numBlocks, _blockSize >> > (_randomState, seed);




}

void CSRAdapter::CalculateMeshGridDerivativeWithEfficacyFinite(const std::vector<inttype>& vecindex,
    const std::vector<fptype>& vecrates, const std::vector<fptype>& efficacy, const std::vector<fptype>& cell_widths, const std::vector<inttype>& cell_offsets, double timestep)
{
    for (inttype m = 0; m < _nr_grid_connections; m++)
    {
        unsigned int mesh_m = vecindex[m];

        if (_group.getGroupObjects()[mesh_m] == 0)
            continue;

        // be careful to use this block size
        inttype numBlocks = (_group._vec_num_objects[mesh_m] + _blockSize - 1) / _blockSize;

        generatePoissonSpikes << <numBlocks, _blockSize >> > (_group._vec_num_objects[mesh_m], _group._vec_num_object_offsets[mesh_m], vecrates[m], timestep, _random_poisson, _randomState);

        CudaCalculateGridDerivativeCsrFinite << <numBlocks, _blockSize >> > (_group._vec_num_objects[mesh_m], _group._vec_num_object_offsets[mesh_m], _random_poisson, _group._vec_objects_to_index,
            _group._vec_objects_refract_times, _group._vec_objects_refract_index,
            _grid_forward_val[m], _grid_forward_ia[m], _grid_forward_ja[m], _offsets[mesh_m], _randomState);

        for (int n = _nr_grid_connections; n < vecrates.size(); n++)
        {
            inttype mat_index = _grid_transforms.size() + (n - _nr_grid_connections);
            unsigned int mesh_n = vecindex[n];

            if (_group.getGroupObjects()[mesh_n] == 0) {
                continue;
            }

            inttype numBlocks = (_group._vec_num_objects[mesh_n] + _blockSize - 1) / _blockSize;
            // be careful to use this block size
            generatePoissonSpikes << <numBlocks, _blockSize >> > (_group._vec_num_objects[mesh_n], _group._vec_num_object_offsets[mesh_n], vecrates[n], timestep, _random_poisson, _randomState);

            CudaUpdateFiniteObjects << <numBlocks, _blockSize >> > (_group._vec_num_objects[mesh_n], _group._vec_num_object_offsets[mesh_n], _random_poisson, _group._vec_objects_to_index,
                _group._vec_objects_refract_times, _group._vec_objects_refract_index, _forward_val[mat_index], _forward_ia[mat_index], _forward_ja[mat_index],
                _group._map, _group._unmap, _offsets[mesh_n], _randomState);

        }
    }
    hipDeviceSynchronize();

}

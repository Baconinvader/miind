#include "hip/hip_runtime.h"
// Copyright (c) 2005 - 2015 Marc de Kamps
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without modification, are permitted provided that the following conditions are met:
//
//    * Redistributions of source code must retain the above copyright notice, this list of conditions and the following disclaimer.
//    * Redistributions in binary form must reproduce the above copyright notice, this list of conditions and the following disclaimer in the documentation
//      and/or other materials provided with the distribution.
//    * Neither the name of the copyright holder nor the names of its contributors may be used to endorse or promote products derived from this software
//      without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
// WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY
// DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF
// USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING
// NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//
//      If you use this software in work leading to a scientific publication, you should include a reference there to
//      the 'currently valid reference', which can be found at http://miind.sourceforge.net

#include <iostream>
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cmath>
#include "CudaEuler.cuh"
#include "CudaOde2DSystemAdapter.cuh"

using namespace CudaTwoDLib;

namespace {
	const float tolerance = 1e-6;
}

#define checkCudaErrors(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
  if (code != hipSuccess) {
    fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
    if (abort) exit(code);
  }
}

CudaOde2DSystemAdapter::CudaOde2DSystemAdapter
(
    TwoDLib::Ode2DSystemGroup& group
):
_group(group),
_time_step(group.MeshObjects()[0].TimeStep()),
_mesh_size(group.MeshObjects().size()),
_n(group.Mass().size()),
_hostmass(_n,0.),
_hostmap(_n,0.),
_offsets(group.Offsets()),
_res_to(group.MeshObjects().size(),0),
_res_from(group.MeshObjects().size(),0),
_res_alpha(group.MeshObjects().size(),0),
_host_fs(group.MeshObjects().size(),0)
{
    this->FillMass();
    this->FillMapData();
    this->FillReversalMap(group.MeshObjects(),group.MapReversal());
    this->FillResetMap(group.MeshObjects(),group.MapReset());
}

void CudaOde2DSystemAdapter::TransferMapData()
{

    for( inttype i = 0; i < _n; i++)
        _hostmap[i] = _group.Map(i);

    checkCudaErrors(hipMemcpy(_map,&_hostmap[0],_n*sizeof(inttype),hipMemcpyHostToDevice));
}

void CudaOde2DSystemAdapter::FillMapData(){
    checkCudaErrors(hipMalloc((inttype**)&_map,_n*sizeof(inttype)));

    this->TransferMapData();
}

void CudaOde2DSystemAdapter::DeleteMass()
{
    hipFree(_mass);
}

void CudaOde2DSystemAdapter::DeleteMapData()
{
    hipFree(_map);
}

CudaOde2DSystemAdapter::~CudaOde2DSystemAdapter()
{
    this->DeleteMass();
    this->DeleteMapData();
    this->DeleteReversalMap();
    this->DeleteResetMap();
}

void CudaOde2DSystemAdapter::FillMass()
{
    checkCudaErrors(hipMalloc((fptype**)&_mass,_n*sizeof(fptype)));

    for(inttype i = 0; i < _n; i++)
        _hostmass[i] = _group.Mass()[i];
    this->Validate();
     checkCudaErrors(hipMemcpy(_mass,&_hostmass[0],_n*sizeof(fptype),hipMemcpyHostToDevice));
}


void CudaOde2DSystemAdapter::Validate() const
{
    // check wether the mass array of the Ode2DSystemGroup has been initialized properly. This means the mass must
    // add up to the number of meshes

    fptype sum = 0.;
    for(int i = 0; i < _n; i++)
       sum += _hostmass[i];

    fptype nmesh = static_cast<fptype>(_group.MeshObjects().size());
    if (fabs(sum - nmesh ) > tolerance){
	fprintf(stderr,"Total mass  unequal to number of mesh objects:%f, %f\n",sum,nmesh);
        exit(0);
    }
}

void CudaOde2DSystemAdapter::Evolve()
{
    _group.Evolve();
    this->TransferMapData();
}

void CudaOde2DSystemAdapter::EvolveWithoutMeshUpdate()
{
    _group.EvolveWithoutMeshUpdate();
		this->TransferMapData();
}

void CudaOde2DSystemAdapter::Dump(const std::vector<std::ostream*>& vec_stream, int mode)
{
     checkCudaErrors(hipMemcpy(&_hostmass[0],_mass,_n*sizeof(fptype),hipMemcpyDeviceToHost));
     for(inttype i = 0; i < _n; i++)
        _group.Mass()[i] = _hostmass[i];
     _group.Dump(vec_stream, mode);
}

const std::vector<fptype>& CudaOde2DSystemAdapter::F() const
{
     checkCudaErrors(hipMemcpy(&_host_fs[0],_fs,_mesh_size*sizeof(fptype),hipMemcpyDeviceToHost));
     for (auto& rate: _host_fs)
         rate /= _time_step;
     return _host_fs;
}

void CudaOde2DSystemAdapter::FillResetMap
(
    const std::vector<TwoDLib::Mesh>& vec_mesh,
    const std::vector<std::vector<TwoDLib::Redistribution> >& vec_vec_reset
)
{
    hipMalloc(&_fs, _mesh_size*sizeof(fptype));
    std::vector<fptype> vec_rates(_mesh_size,0.);
    checkCudaErrors(hipMemcpy(_fs,&vec_rates[0],_mesh_size*sizeof(fptype),hipMemcpyHostToDevice));

   for(inttype m = 0; m < _mesh_size; m++)
   {
       _nr_resets.push_back(vec_vec_reset[m].size());
       checkCudaErrors(hipMalloc((inttype**)&_res_to[m],   vec_vec_reset[m].size()*sizeof(inttype)));
       checkCudaErrors(hipMalloc((inttype**)&_res_from[m], vec_vec_reset[m].size()*sizeof(inttype)));
       checkCudaErrors(hipMalloc((fptype**)&_res_alpha[m], vec_vec_reset[m].size()*sizeof(fptype)));
       std::vector<inttype> vec_to;
       std::vector<inttype> vec_from;
       std::vector<fptype>  vec_alpha;
       for(inttype i = 0; i < vec_vec_reset[m].size(); i++)
       {
           vec_to.push_back(_group.Map(m,vec_vec_reset[m][i]._to[0],  vec_vec_reset[m][i]._to[1]));
           vec_from.push_back(_group.Map(m,vec_vec_reset[m][i]._from[0],vec_vec_reset[m][i]._from[1]));
           vec_alpha.push_back(vec_vec_reset[m][i]._alpha);
       }
       checkCudaErrors(hipMemcpy(_res_to[m],&vec_to[0],vec_to.size()*sizeof(inttype),hipMemcpyHostToDevice));
       checkCudaErrors(hipMemcpy(_res_from[m],&vec_from[0],vec_from.size()*sizeof(inttype),hipMemcpyHostToDevice));
       checkCudaErrors(hipMemcpy(_res_alpha[m],&vec_alpha[0],vec_alpha.size()*sizeof(fptype),hipMemcpyHostToDevice));
  }
}

void CudaOde2DSystemAdapter::RedistributeProbability()
{
    for (inttype m = 0; m < _mesh_size; m++){
        fptype* f = _fs+m;
        MapReset<<<1,1>>>(_nr_resets[m],_res_from[m],_res_to[m],_res_alpha[m],_mass,_map,f);
    }
}

void CudaOde2DSystemAdapter::MapFinish()
{
    for (inttype m = 0; m < _mesh_size; m++)
        ResetFinish<<<1,1>>>(_nr_resets[m],_res_from[m],_mass,_map);
}

void CudaOde2DSystemAdapter::FillReversalMap
(
    const std::vector<TwoDLib::Mesh>& vec_mesh,
    const std::vector<std::vector<TwoDLib::Redistribution> >& vec_vec_reversal
)
{
     _n_rev = 0;
     for(inttype m = 0; m < vec_mesh.size(); m++)
         _n_rev += vec_vec_reversal[m].size();

     hipMallocManaged(&_rev_to,    _n_rev*sizeof(inttype));
     hipMallocManaged(&_rev_from,  _n_rev*sizeof(inttype));
     hipMallocManaged(&_rev_alpha, _n_rev*sizeof(fptype));

     inttype index = 0;
     for(inttype m = 0; m < vec_mesh.size(); m++){
          for( const TwoDLib::Redistribution& r: vec_vec_reversal[m] ){
              _rev_to[index]   = _group.Map(m,r._to[0],r._to[1]);
              _rev_from[index] = _group.Map(m,r._from[0],r._from[1]);
              _rev_alpha[index] = r._alpha;
              index++;
          }
     }
}

void CudaOde2DSystemAdapter::RemapReversal()
{
    MapReversal<<<1,1>>>(_n_rev, _rev_from, _rev_to, _rev_alpha, _mass, _map);
}


void CudaOde2DSystemAdapter::DeleteResetMap()
{
    hipFree(_fs);

    for(inttype m = 0; m < _mesh_size; m++)
    {
	hipFree(_res_to[m]);
        hipFree(_res_from[m]);
        hipFree(_res_alpha[m]);
    }

}

void CudaOde2DSystemAdapter::DeleteReversalMap()
{
    hipFree(_rev_to);
    hipFree(_rev_from);
    hipFree(_rev_alpha);

}
